#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void calc_GPU(int *vect_a, double *vect_result){
	int thr = threadIdx.x;
	int blk = blockIdx.x;
	int size = 4;
	__shared__ double temp[4];
	double price[4] = {29.99, 14.99, 9.99, 24.99};
	temp[thr] = vect_a[blk*blockDim.x + thr]*price[thr];
	__syncthreads();
	for (int i = 0; i < size; i++)
		vect_result[blk] += temp[i];
}

int main(void) {
	int size = 4;
	int a[7*size] = {3,5,2,0, 2,4,5,1, 0,3,3,1, 3,5,4,4, 4,5,5,3, 10,13,21,16, 8,11,15,8};
	double result[7] = {0,0,0,0,0,0,0};
	
	int *vect_a;
	double *vect_result;
	hipMalloc((void**) &vect_a, sizeof(int)*7*size);
	hipMalloc((void**) &vect_result, sizeof(double)*7);
	
	hipMemcpy(vect_a, &a, sizeof(int)*7*size, hipMemcpyHostToDevice);

	calc_GPU<<<7,size>>>(vect_a, vect_result);
	hipDeviceSynchronize();

	hipMemcpy(&result, vect_result, sizeof(double)*7, hipMemcpyDeviceToHost);
	hipFree(vect_a);
	hipFree(vect_result);

	printf("Sales: %.2f %.2f %.2f %.2f %.2f %.2f %.2f\n", result[0], result[1], 
result[2], result[3], result[4], result[5], result[6]);
	return 0;
}
