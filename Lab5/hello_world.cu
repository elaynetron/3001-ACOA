
#include <hip/hip_runtime.h>
#include <stdio.h>
__global__ void hello_GPU(int num) {
	int thr = threadIdx.x;
	printf("Hello from GPU%d[%d]!\n", num, thr);
}	

int main(void){
	printf("Hello from CPU!\n");
	hello_GPU<<<1,4>>>(1);
	hello_GPU<<<1,6>>>(2);
	hipDeviceSynchronize();
	return 0;
}
