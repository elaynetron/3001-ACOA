#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add_GPU(int *vect_a, int *vect_b, int *vect_c){
	int thr = threadIdx.x;
	vect_c[thr] = vect_a[thr] + vect_b[thr];
}

int main(void) {
	int size = 4;
	int a[size] = {22,13,16,5};
	int b[size] = {5,22,17,37};
	int c[size];
	
	int *vect_a, *vect_b, *vect_c;
	hipMalloc((void**) &vect_a, sizeof(int)*size);
	hipMalloc((void**) &vect_b, sizeof(int)*size);
	hipMalloc((void**) &vect_c, sizeof(int)*size);
	
	hipMemcpy(vect_a, &a, sizeof(int)*size, hipMemcpyHostToDevice);
	hipMemcpy(vect_b, &b, sizeof(int)*size, hipMemcpyHostToDevice);

	add_GPU<<<1,size>>>(vect_a, vect_b, vect_c);

	hipMemcpy(&c, vect_c, sizeof(int)*size, hipMemcpyDeviceToHost);
	hipFree(vect_a);
	hipFree(vect_b);
	hipFree(vect_c);

	printf("C: %d %d %d %d\n", c[0], c[1], c[2], c[3]);
	return 0;
}
