#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void dotprod_GPU(int *vect_a, int *vect_b, int *vect_c){
	int thr = threadIdx.x;
	int size = 4;
	__shared__ int temp[4];
	temp[thr] = vect_a[thr] * vect_b[thr];
	__syncthreads();

	if (thr == 0){
		int total = 0;
		for (int i = 0; i < size; i++)
			total += temp[i];
		*vect_c = total;
	}
}

int main(void){
	int size = 4;
	int a[size] = {22,13,16,5}; 
	int b[size] = {5,22,17,37}; 
	int c;

	int *vect_a, *vect_b, *vect_c;
	hipMalloc((void**)&vect_a, sizeof(int)*size);
	hipMalloc((void**)&vect_b, sizeof(int)*size);
	hipMalloc((void**)&vect_c, sizeof(int));

	hipMemcpy(vect_a, &a, sizeof(int)*size, hipMemcpyHostToDevice); 
	hipMemcpy(vect_b, &b, sizeof(int)*size, hipMemcpyHostToDevice);
	
	dotprod_GPU<<<1,size>>>(vect_a, vect_b, vect_c);
	hipMemcpy(&c, vect_c, sizeof(int), hipMemcpyDeviceToHost); 
	hipFree(vect_a);
	hipFree(vect_b);
	hipFree(vect_c);

	printf("Answer = %d\n", c);
	return 0;
}
